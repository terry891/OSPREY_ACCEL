#include "hip/hip_runtime.h"
//
// Created by Chris Kjellqvist on 11/9/22.
//

#include <cstdlib>
#include <cstring>
#include <cassert>
#include <random>
#include <chrono>
#include <iostream>
#include "gemm_cpu.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"


//#define UNIT

#ifdef UNIT
int mat_dim = 32;
#else
int mat_dim = 1 << 14;
#endif

inline volatile void flush_line(void *ptr) {
  asm volatile ("clflush (%0)"::"r" (ptr));
}

void flush_segment(char *ptr, size_t length) {
  assert((length & 0x3F) == 0);
  for (int i = 0; i < length >> 6; ++i) {
    flush_line(ptr);
    ptr += 64;
  }
}


int n_cores = 2;
pthread_mutex_t time_lock = PTHREAD_MUTEX_INITIALIZER;
unsigned long long cpytime = 0, addrtime = 0, exectime = 0, cpybacktime = 0;
pthread_t *threads;

unsigned long long sum = 0;

int main() {
  int max_mag = 100;
  // initialize c++ random number generator
  std::random_device rd;
  std::mt19937 gen(177);
  size_t mat_size = mat_dim * mat_dim * sizeof(float);
  auto a = (float *) malloc(mat_size);
  auto b = (float *) malloc(mat_size);
  auto c = (float *) malloc(mat_size);
  auto correct = (float *) malloc(mat_size);

  memset(c, 0, sizeof(float) * mat_dim * mat_dim);
  std::uniform_int_distribution<> dis(0, max_mag);

  // allocate two mat_dimxmat_dim matrices of signed ints
  // randomly initialize these matrices
  for (int i = 0; i < mat_dim * mat_dim; ++i) {
    a[i] = dis(gen);
    b[i] = dis(gen);
  }

  hipError_t cudaStat;
  hipblasStatus_t stat;
  hipblasHandle_t handle;
  float constant1 = 1;

  float *devPtrA, *devPtrB, *devPtrC;
  cudaStat = hipMalloc((void **) &devPtrA, mat_dim * mat_dim * sizeof(*a));
  if (cudaStat != hipSuccess) {
    printf("device memory allocation failed");
    return EXIT_FAILURE;
  }

  cudaStat = hipMalloc((void **) &devPtrB, mat_dim * mat_dim * sizeof(*a));
  if (cudaStat != hipSuccess) {
    printf("device memory allocation failed");
    hipFree(devPtrA);
    return EXIT_FAILURE;
  }

  cudaStat = hipMalloc((void **) &devPtrC, mat_dim * mat_dim * sizeof(*a));
  if (cudaStat != hipSuccess) {
    printf("device memory allocation failed");
    hipFree(devPtrA);
    hipFree(devPtrB);
    return EXIT_FAILURE;
  }

  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("CUBLAS initialization failed\n");
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    return EXIT_FAILURE;
  }

  stat = hipblasSetMatrix(mat_dim, mat_dim, sizeof(*a), b, mat_dim, devPtrB, mat_dim);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  stat = hipblasSetMatrix(mat_dim, mat_dim, sizeof(*a), c, mat_dim, devPtrC, mat_dim);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  stat = hipblasSetMatrix(mat_dim, mat_dim, sizeof(*a), a, mat_dim, devPtrA, mat_dim);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data download failed");
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  // hipblasStatus_t hipblasSgemm(hipblasHandle_t handle,
  //                           hipblasOperation_t transa, hipblasOperation_t transb,
  //                           int m, int n, int k,
  //                           const float           *alpha,
  //                           const float           *A, int lda,
  //                           const float           *B, int ldb,
  //                           const float           *beta,
  //                           float           *C, int ldc)


  // compute gemm
  int times = 1;
  gemm_cpu<float, 8>(a, b, correct, mat_dim);
  for (int i = 0; i < mat_dim * mat_dim; ++i) {
    correct[i] *= times;
  }
  unsigned long long ttime = 0;
  for (int i = 0; i < times; ++i) {
    auto start = std::chrono::system_clock::now();
    stat = hipblasSgemm(handle,
                       HIPBLAS_OP_N,
                       HIPBLAS_OP_N,
                       mat_dim,
                       mat_dim,
                       mat_dim,
                       &constant1,
                       devPtrA, mat_dim,
                       devPtrB, mat_dim,
                       &constant1,
                       devPtrC, mat_dim);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
      printf("data download failed");
      hipFree(devPtrA);
      hipFree(devPtrB);
      hipFree(devPtrC);
      hipblasDestroy(handle);
      return EXIT_FAILURE;
    }

    auto end = std::chrono::system_clock::now();
    ttime += std::chrono::duration_cast<std::chrono::microseconds>(end - start).count();
  }


  stat = hipblasGetMatrix(mat_dim, mat_dim, sizeof(*a), devPtrC, mat_dim, c, mat_dim);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    printf("data upload failed");
    hipFree(devPtrA);
    hipFree(devPtrB);
    hipFree(devPtrC);
    hipblasDestroy(handle);
    return EXIT_FAILURE;
  }

  hipFree(devPtrA);
  hipFree(devPtrB);
  hipFree(devPtrC);
  hipblasDestroy(handle);

  std::cout << "GPU GeMM - Took " << ttime << "us. Avg: " << (ttime / times) << "us." << std::endl;
  std::cout << sum << std::endl;


  for (int i = 0; i < mat_dim * mat_dim; ++i) {
    auto pdiff = abs(correct[i] - c[i]) / abs(correct[i]);
    if (pdiff > 0.01) {
      std::cerr << "Incorrect result: " << i << " " << pdiff << " " << correct[i] << " " << c[i] << std::endl;
      return EXIT_FAILURE;
    }
  }





}
